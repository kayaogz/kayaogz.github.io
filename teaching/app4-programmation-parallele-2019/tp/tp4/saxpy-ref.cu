#include <iostream>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void saxpy(size_t n, float a, float *x, float *y)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) y[i] = a * x[i] + y[i];
}

int main(int argc, char **argv)
{
  size_t N = atoi(argv[1]);
  float *x, *y, *res, *dx, *dy;
  float a = 2.0f;

  // Allocate and initialize vectors x and y on the CPU
  x = (float *) malloc(N * sizeof(float));
  y = (float *) malloc(N * sizeof(float));
  res = (float *) malloc(N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = i;
    y[i] = 1.0f;
  }

  // Allocate device vectors dx and dy, then copy x and y into them
  hipMalloc(&dx,  N * sizeof(float));
  hipMalloc(&dy, N * sizeof(float));

  hipMemcpy(dx, x, N * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dy, y, N * sizeof(float), hipMemcpyHostToDevice);

  // Launch the CUDA kernel for saxpy
  int blockSize = 1024;
  saxpy<<<(N + blockSize - 1) / blockSize, blockSize>>>(N, a, dx, dy);

  auto begin = std::chrono::high_resolution_clock::now(); 
  hipMemcpy(res, dy, N * sizeof(float), hipMemcpyDeviceToHost);

  // Verify results
  {
    int i;
    for (i = 0; i < N; i++) {
      float temp = a * x[i] + y[i];
      if (std::abs(res[i] - temp) / std::max(1e-6f, temp) > 1e-6) { 
        cout << res[i] << " " << temp << endl;
        break;
      }
    }
    if (i == N) {
      cout << "saxpy on GPU is correct." << endl;
    } else {
      cout << "saxpy on GPU is incorrect on element " << i << "." << endl;
    }
  }

  return 0;
}
