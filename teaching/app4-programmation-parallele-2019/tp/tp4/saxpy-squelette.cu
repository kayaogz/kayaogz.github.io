#include <iostream>
#include <algorithm>
#include <chrono>
#include <hip/hip_runtime.h>

using namespace std;

__global__
void saxpy(size_t n, float alpha, float *a, float *b)
{
}

int main(int argc, char **argv)
{
  size_t N = 1024;
  float *a, *b, *res, *da, *db;
  float alpha = 2.0f;

  // Allocate and initialize vectors a and b on the CPU
  a = (float *) malloc(N * sizeof(float));
  b = (float *) malloc(N * sizeof(float));
  res = (float *) malloc(N * sizeof(float));

  for (int i = 0; i < N; i++) {
    a[i] = i;
    b[i] = 1.0f;
  }

  // Allocate device vectors da and db, then copy a and b into them
  // A FAIRE

  // Launch the CUDA kernel for saxpy
  // A FAIRE

  // Copy results back to the CPU
  // A FAIRE

  // Verify results
  {
    int i;
    for (i = 0; i < N; i++) {
      float temp = alpha * a[i] + b[i];
      if (std::abs(res[i] - temp) / std::max(1e-6f, temp) > 1e-6) { 
        cout << res[i] << " " << temp << endl;
        break;
      }
    }
    if (i == N) {
      cout << "saxpy on GPU is correct." << endl;
    } else {
      cout << "saxpy on GPU is incorrect on element " << i << "." << endl;
    }
  }

  return 0;
}
